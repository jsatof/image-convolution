#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <opencv2/opencv.hpp>

std::string get_image_name(std::string arg);
void image_to_matrix(cv::Mat image, int *blue, int *green, int *red);
void matrix_to_image(cv::Mat image, int *blue, int *green, int *red);

__global__ void convolve(int *input, int *output, int width, double *kernel, int kernel_length) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int k = kernel_length / 2;
    double sum = 0.0;

    for(int i = -k; i <= k; i++) {
        for(int j = -k; j <= k; j++) {
            double kernel_value = kernel[(i + k) * kernel_length + (j + k)];
            int matrix_value = input[(i + x) * width + (j + y)];

            sum += matrix_value * kernel_value;
        }
    }

    output[x * width + y] = sum;
}

int main(int argc, char **argv) {
    if(argc != 2) {
        printf("usage: ./run <name of image file>\n");
        return 1;
    }

    std::string filename = get_image_name(argv[1]);

    if(filename.compare("invalid") == 0) {
        printf("Invalid Image.\n");
        return 1;
    }

    cv::Mat image = cv::imread(filename);
    int height = image.rows;
    int width = image.cols;

    size_t bytes_image = height * width * sizeof(int);
    int *h_blue = (int*) malloc(bytes_image);
    int *h_green = (int*) malloc(bytes_image);
    int *h_red = (int*) malloc(bytes_image);
    image_to_matrix(image, h_blue, h_green, h_red);

    int kernel_length = 3;
    double h_kernel[9] = {
        0.0, -1.0, 0.0,
        -1.0, 5.0, -1.0,
        0.0, -1.0, 0.0
    };

    double *d_kernel;
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    int *d_blue;
    hipMalloc(&d_blue, bytes_image);
    hipMemcpy(d_blue, h_blue, bytes_image, hipMemcpyHostToDevice);
    int *d_green;
    hipMalloc(&d_green, bytes_image);
    hipMemcpy(d_green, h_green, bytes_image, hipMemcpyHostToDevice);
    int *d_red;
    hipMalloc(&d_red, bytes_image);
    hipMemcpy(d_red, h_red, bytes_image, hipMemcpyHostToDevice);

    int *d_result_blue;
    hipMalloc(&d_result_blue, bytes_image);
    int *d_result_green;
    hipMalloc(&d_result_green, bytes_image);
    int *d_result_red;
    hipMalloc(&d_result_red, bytes_image);

    int num_threads = 16;
    dim3 threads(num_threads, num_threads);
    dim3 blocks(width / num_threads + 1, height / num_threads + 1);

    long start_time = clock();
    convolve <<< blocks, threads >>> (d_blue, d_result_blue, width, d_kernel, kernel_length);
    convolve <<< blocks, threads >>> (d_green, d_result_green, width, d_kernel, kernel_length);
    convolve <<< blocks, threads >>> (d_red, d_result_red, width, d_kernel, kernel_length);
    long end_time = clock();

    int *h_result_blue = (int*) malloc(bytes_image);
    hipMemcpy(h_result_blue, d_result_blue, bytes_image, hipMemcpyDeviceToHost);
    int *h_result_green = (int*) malloc(bytes_image);
    hipMemcpy(h_result_green, d_result_green, bytes_image, hipMemcpyDeviceToHost);
    int *h_result_red = (int*) malloc(bytes_image);
    hipMemcpy(h_result_red, d_result_red, bytes_image, hipMemcpyDeviceToHost);
 
    matrix_to_image(image, h_result_blue, h_result_green, h_result_red);
    cv::imwrite("output_cuda.jpg", image);

    double conv_time = (double) (end_time - start_time) / CLOCKS_PER_SEC;
    printf("Convolution Time: %fs\n", conv_time);

    free(h_blue);
    free(h_green);
    free(h_red);
    free(h_result_blue);
    free(h_result_green);
    free(h_result_red);
    hipFree(d_blue);
    hipFree(d_green);
    hipFree(d_red);
    hipFree(d_result_blue);
    hipFree(d_result_green);
    hipFree(d_result_red);
    hipFree(d_kernel);

    return 0;
}

std::string get_image_name(std::string arg) {
    if(arg.compare("harold.jpg") == 0 || arg.compare("harold") == 0) {
        return "../images/harold.jpg";
    } 
    if(arg.compare("misha_mansoor.jpg") == 0 || arg.compare("misha") == 0) {
        return "../images/misha_mansoor.jpg";
    }
    if(arg.compare("christmas.jpg") == 0 || arg.compare("xmas") == 0) {
        return "../images/christmas.jpg";
    }
    if(arg.compare("nier.jpg") == 0 || arg.compare("nier") == 0) {
        return "../images/nier.jpg";
    }
    return "invalid";
}

void image_to_matrix(cv::Mat image, int *blue, int *green, int *red) {
    for(int i = 0; i < image.rows; i++) {
        for(int j = 0; j < image.cols; j++) {
            blue[i * image.cols + j] = image.at<cv::Vec3b>(i, j)[0];
            green[i * image.cols + j] = image.at<cv::Vec3b>(i, j)[1];
            red[i * image.cols + j] = image.at<cv::Vec3b>(i, j)[2];
        }
    }
}

void matrix_to_image(cv::Mat image, int *blue, int *green, int *red) {
    for(int i = 0; i < image.rows; i++) {
        for(int j = 0; j < image.cols; j++) {
            image.at<cv::Vec3b>(i, j)[0] = blue[i * image.cols + j];
            image.at<cv::Vec3b>(i, j)[1] = green[i * image.cols + j];
            image.at<cv::Vec3b>(i, j)[2] = red[i * image.cols + j];
        }
    }
}